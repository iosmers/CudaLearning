#include <hip/hip_runtime.h>
#include <iostream>

// Kernel: 块内使用共享内存进行归约求和
__global__ void reduce_sum_kernel(const float* input, float* output, int size) {
    extern __shared__ float sdata[]; // 拷贝到共享内存的目的是为了加快访问速度

    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // 拷贝数据到共享内存
    sdata[tid] = (idx < size) ? input[idx] : 0.0f;
    __syncthreads();

    // 块内规约：对半加法
    for (int s = blockDim.x / 2; s > 0; s = s/2) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        // 必须要加同步
        __syncthreads();
    }

    // 每个 block 的第一个线程写回结果
    if (tid == 0) {
        output[blockIdx.x] = sdata[0];
    }
}

// Host接口：多步规约完成总和
void reduce_sum(const float* h_input, float* h_output, int size) {
    float *d_input, *d_intermediate;
    int threads_per_block = 256;
    int blocks = (size + threads_per_block - 1) / threads_per_block;

    hipMalloc(&d_input, sizeof(float) * size);
    hipMemcpy(d_input, h_input, sizeof(float) * size, hipMemcpyHostToDevice);

    hipMalloc(&d_intermediate, sizeof(float) * blocks);

    // 第一步规约
    reduce_sum_kernel<<<blocks, threads_per_block, threads_per_block * sizeof(float)>>>(
        d_input, d_intermediate, size);

    // 复制中间结果回 Host 并求最终和
    float* h_intermediate = new float[blocks];
    hipMemcpy(h_intermediate, d_intermediate, sizeof(float) * blocks, hipMemcpyDeviceToHost);

    float final_sum = 0.0f;
    for (int i = 0; i < blocks; ++i) {
        final_sum += h_intermediate[i];
    }

    *h_output = final_sum;

    delete[] h_intermediate;
    hipFree(d_input);
    hipFree(d_intermediate);
}

// 测试用例
int main() {
    const int size = 1024;
    float h_input[size];
    for (int i = 0; i < size; ++i) h_input[i] = 1.0f; // 所有元素设为1，预期总和为1024

    float h_output = 0.0f;
    reduce_sum(h_input, &h_output, size);

    std::cout << "Reduce sum result: " << h_output << std::endl;
    return 0;
}
